﻿#include "hip/hip_runtime.h"

#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdexcept>


void handle_cuda_result(hipError_t cuerr, char msg[]) {
    if (cuerr != hipSuccess) {
        fprintf(stderr, hipGetErrorString(cuerr));
        fprintf(stderr, "\n");
       
    }
}

void upload_to_device(hipArray** hipArray, hipChannelFormatDesc& channelDesc, int* in, int n, int m) {
    hipError_t cuerr;
    cuerr = hipMallocArray(hipArray, &channelDesc, m, n);
    handle_cuda_result(cuerr, "Cannot allocate memory for array");
    cuerr = hipMemcpyToArray(*hipArray, 0, 0, in, sizeof(int) * n * m, hipMemcpyHostToDevice);
    handle_cuda_result(cuerr, "Cannot copy to device");
}

void allocate_in_device(int** resultGpuPointer, int size) {
    int sizeInBytes = size * sizeof(int);
    // выделяем память
    hipError_t cuerr = hipMalloc((void**)resultGpuPointer, sizeInBytes);
    handle_cuda_result(cuerr, "Cannot allocate device array");
}

void download_from_device(int* gpuMatPointer, int* resultMat, int size) {
    int sizeInBytes = size * sizeof(int);
    // копируем массив
    hipError_t cuerr = hipMemcpy(resultMat, gpuMatPointer, sizeInBytes, hipMemcpyDeviceToHost);
    handle_cuda_result(cuerr, "Cannot copy a array from device to host");
}